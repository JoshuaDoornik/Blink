#include "hip/hip_runtime.h"
#include <Python.h>
#include <stdio.h>

extern "C" __global__ void helloFromGPU();

static PyObject *SpamError;

static PyObject * hello_gpu(PyObject *self, PyObject *args)
{
    helloFromGPU <<<1,10>>>();
    hipDeviceReset();
    return Py_None;
}

__global__ void helloFromGPU(){
    printf("Hello world, from my GPU! \n");
}

static PyMethodDef SpamMethods[] = {
    {"hello_gpu",  hello_gpu, METH_VARARGS,
     "say hello world 10 times parallel from your GPU. because why not?."},
    {NULL, NULL, 0, NULL}        /* Sentinel */
};


PyMODINIT_FUNC inithelloworld(void)
{
    PyObject *m;

    m = Py_InitModule("helloworld", SpamMethods);
    if (m == NULL)
        return;

    SpamError = PyErr_NewException("gpu.error", NULL, NULL);
    Py_INCREF(SpamError);
}