#include "hip/hip_runtime.h"
#include <Python.h>
#include <stdio.h>

extern "C" __global__ void helloFromGPU();
extern "C" __global__ void mapFromGPU();
static PyObject *SpamError;

static PyObject * hello_gpu(PyObject *self, PyObject *args)
{
    helloFromGPU <<<1,10>>>();
    hipDeviceReset();
    return Py_RETURN_NONE;
}



static PyObject * map(PyObject *self, PyObject *args)
{
    PyObject * listObj;
    PyObject *temp;
    //The input arguments come as a tuple, we parse the args to get the various variables
    //In this case it's only one list variable, which will now be referenced by listObj
    if (! PyArg_ParseTuple( args, "OO:blink_map", &listObj, &temp)){
        return NULL;
    }
    if (!PyCallable_Check(temp)) {
        PyErr_SetString(PyExc_TypeError, "parameter must be callable");
        return NULL;
    }
    helloFromGPU <<<1,10>>>();
    hipDeviceReset();
    return Py_RETURN_NONE;
}

static PyMethodDef SpamMethods[] = {
    {"hello_gpu",  hello_gpu, METH_VARARGS,
     "say hello world 10 times parallel from your GPU. because why not?."},
     {"map",  map, METH_VARARGS,
     "map a function to a datastructure (for now a list) using your GPU for more parallel execution."},
    {NULL, NULL, 0, NULL}        /* Sentinel */
};

__global__ void helloFromGPU(){
    printf("Hello world, from my GPU! \n");
}

__global__ void mapFromGPU(PyObject* Pylist, PyObject* callable){
    printf("Hello world, from my GPU! \n");
}

PyMODINIT_FUNC inithelloworld(void)
{
    PyObject *m;

    m = Py_InitModule("helloworld", SpamMethods);
    if (m == NULL)
        return;


    // figure out what you're using for future grid sizes
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
    hipDeviceProp_t deviceProp;
    for(int i = 0; i < deviceCount;i++){
        hipGetDeviceProperties(&deviceProp, i);
        printf("[+] located Device %d: %s\n", i, deviceProp.name);
        printf("[+] set warpsize to %d\n",deviceProp.warpSize);
    }
    printf("\n");
    SpamError = PyErr_NewException("gpu.error", NULL, NULL);
    Py_INCREF(SpamError);
}