#include "hip/hip_runtime.h"
#include <python.h>

//python boiler plate. define methods with extern "C" so the compiler doesnt prank us.
extern "C" static PyObject *access_map(int (*fp)(int), int[] arg. int size));
__global__ void map(int[] arg, int (*fp)(int)){
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    arg[i] = fp(arg[i]);

}

//access means that the python part should be accessing this part of the code and no other functions
PyObject *access_map(int (*fp)(int), int[] arr, int size)){
    int threads = (size >= 256) ? 256 : size; 
    int blocks = size/256 + 1;
    int * d_arr;
    //allocating and copying. this takes about 600 clock cycles, but theres no alternative.
    hipMalloc(&d_arr, size*sizeof(int)); 
    hipMemcpy(d_arr, arr, size*sizeof(int),hipMemcpyHostToDevice);
    map<<<blocks,threads>>>(d_x,d_arr);
    hipMemcpy(arr,d_arr, size*sizeof(int), hipMemcpyDeviceToHost); 
    hipFree(d_arr);
}