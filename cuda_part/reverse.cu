#include "hip/hip_runtime.h"
#include <stdio.h>
#include <python.h>

//python boiler plate. define methods with extern "C" so the compiler doesnt prank us.
extern "C" static PyObject *access_swap(PyObject *self, PyObject *args);
extern "C" void init_CU_AddVector() ;

/*link method with needed info. <function-name in python module>, <actual-function>,
  <type-of-args the function expects>, <docstring associated with the function>
  */
static PyMethodDef addList_funcs[] = {
    {"cuda_reverse_array", (PyCFunction)access_swap, METH_VARARGS, reverseList_docs},
    {NULL, NULL, 0, NULL}
};
//doc string, to add documentation to our method
static char reverseList_docs[] =
        "reverse the list\n";

PyMODINIT_FUNC initaddList(void){
    Py_InitModule3("cuda_adder", addList_funcs,
                   "Add all ze lists");
}

__global__
void swap(int *a,int size){
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int temp = a[i];
    a[i] = a[size-i];
    a[size-i] = temp;
}

PyObject * access_swap(PyObject *self, PyObject *args){
    PyListObject *to_swap;
    if (!PyArg_ParseTuple(args, "O!", &PyListObject, &to_swap)){  
        return NULL;
    }   
    
    int size = PyList_Size(&to_swap);
    int blocks = size/256 + 1;
    int threads= (size >= 256) ? 256 : size; 

    hipMemcpy(d_x, x, size*sizeof(int),hipMemcpyHostToDevice);
    swap<<<blocks,threads>>>(d_x,size-1);
    hipMemcpy(x,d_x, size*sizeof(int), hipMemcpyDeviceToHost); 
    hipFree(d_x);
    free(x);
}

