#include <stdio.h>
#include <hip/hip_runtime.h>

__global__
void swap(int *a,int size){
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int temp = a[i];
    a[i] = a[size-i];
    a[size-i] = temp;
}

int main(void){
    int size = 100;
    int blocks = size/256 + 1;
    int threads= (size >= 256) ? 256 : size; 
    int *x, *d_x;
    x = (int *)malloc(size*sizeof(int));
    hipMalloc(&d_x, size*sizeof(int));
    
    for(int i = 0; i < size; i++){
        x[i] = i;
   }
    printf("%d \n",blocks);    
    hipMemcpy(d_x, x, size*sizeof(int),hipMemcpyHostToDevice);
    swap<<<blocks,threads>>>(d_x,size-1);
    hipMemcpy(x,d_x, size*sizeof(int), hipMemcpyDeviceToHost); 
    for(int i = 0; i < size; i++){
        printf(" element %d = %d\n", i, x[i]);
    }
    hipFree(d_x);
    free(x);
}

