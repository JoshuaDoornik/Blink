#include <stdio.h>
#include <hip/hip_runtime.h>

__global__
void swap(int *a,int size){
    //int size = 255;
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int temp = a[i];
    a[i] = a[size-i];
    a[size-i] = temp;
}

int main(void){
    int size = 10;
    int blocks = size/256 + 1;
    int threads= (size >= 256) ? 256 : size; 
    int *x, *d_x;
    x = (int *)malloc(size*sizeof(int));
    hipMalloc(&d_x, size*sizeof(int));
    
    for(int i = 0; i < size; i++){
        x[i] = i;
}
    
    hipMemcpy(d_x, x, size*sizeof(int),hipMemcpyHostToDevice);
    swap<<<size,blocks>>>(d_x,size);
    hipMemcpy(x,d_x, size*sizeof(int), hipMemcpyDeviceToHost);

    for(int i = 0; i < size; i++){
        printf(" element %d = %d\n", i, x[i]);
}
    hipFree(d_x);
    free(x);
}

